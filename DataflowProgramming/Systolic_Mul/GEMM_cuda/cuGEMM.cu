#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

#define CHECK_CUDA(call) \
    { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            std::cerr << "CUDA error: " << hipGetErrorString(err) << " at " << __FILE__ << ":" << __LINE__ << std::endl; \
            exit(err); \
        } \
    }

#define CHECK_CUBLAS(call) \
    { \
        hipblasStatus_t status = call; \
        if (status != HIPBLAS_STATUS_SUCCESS) { \
            std::cerr << "cuBLAS error at " << __FILE__ << ":" << __LINE__ << std::endl; \
            exit(EXIT_FAILURE); \
        } \
    }

void generateRandomMatrix(float* matrix, int rows, int cols) {
    for (int i = 0; i < rows * cols; ++i) {
        matrix[i] = static_cast<float>(rand()) / RAND_MAX;
    }
}

int main() {
    // const int N = 34500;
    const int N = 34500;
    const float alpha = 1.0f;
    const float beta = 0.0f;

    // Allocate host memory for matrices A, B, and C
    std::vector<float> h_A(N * N);
    std::vector<float> h_B(N * N);
    std::vector<float> h_C(N * N);

    // Generate random values for A and B
    generateRandomMatrix(h_A.data(), N, N);
    generateRandomMatrix(h_B.data(), N, N);

    float* d_A, * d_B, * d_C;
    size_t matrixSize = N * N * sizeof(float);

    // Allocate device memory for matrices A, B, and C
    CHECK_CUDA(hipMalloc(&d_A, matrixSize));
    CHECK_CUDA(hipMalloc(&d_B, matrixSize));
    CHECK_CUDA(hipMalloc(&d_C, matrixSize));

    // Copy matrices A and B to device memory
    CHECK_CUDA(hipMemcpy(d_A, h_A.data(), matrixSize, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_B, h_B.data(), matrixSize, hipMemcpyHostToDevice));

    // Create cuBLAS handle
    hipblasHandle_t handle;
    CHECK_CUBLAS(hipblasCreate(&handle));

    // Set cuBLAS to use Tensor Cores
    CHECK_CUBLAS(hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH));

    // Start GPU timer
    hipEvent_t start, stop;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));
    CHECK_CUDA(hipEventRecord(start, 0));

    // Perform matrix multiplication: C = alpha * A * B + beta * C
    CHECK_CUBLAS(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, d_A, N, d_B, N, &beta, d_C, N));

    // Stop GPU timer
    CHECK_CUDA(hipEventRecord(stop, 0));
    CHECK_CUDA(hipEventSynchronize(stop));
    float milliseconds = 0;
    CHECK_CUDA(hipEventElapsedTime(&milliseconds, start, stop));

    // Copy result matrix C back to host memory
    CHECK_CUDA(hipMemcpy(h_C.data(), d_C, matrixSize, hipMemcpyDeviceToHost));

    // Clean up resources
    CHECK_CUDA(hipFree(d_A));
    CHECK_CUDA(hipFree(d_B));
    CHECK_CUDA(hipFree(d_C));
    CHECK_CUBLAS(hipblasDestroy(handle));
    CHECK_CUDA(hipEventDestroy(start));
    CHECK_CUDA(hipEventDestroy(stop));

    std::cout << "Matrix multiplication completed successfully in " << milliseconds << " ms." << std::endl;
    return 0;
}
