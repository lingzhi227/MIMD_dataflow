﻿#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

#define CHECK_CUDA(call) \
    { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            std::cerr << "CUDA error: " << hipGetErrorString(err) << " at " << __FILE__ << ":" << __LINE__ << std::endl; \
            exit(err); \
        } \
    }

#define CHECK_CUBLAS(call) \
    { \
        hipblasStatus_t status = call; \
        if (status != HIPBLAS_STATUS_SUCCESS) { \
            std::cerr << "cuBLAS error at " << __FILE__ << ":" << __LINE__ << std::endl; \
            exit(EXIT_FAILURE); \
        } \
    }

void generateRandomMatrix(float* matrix, int rows, int cols) {
    for (int i = 0; i < rows * cols; ++i) {
        matrix[i] = static_cast<float>(rand()) / RAND_MAX;
    }
}

int main() {
    const int N = 36600;
    const float alpha = 1.0f;
    const float beta = 0.0f;

    // Allocate host memory for matrices A, B, and C
    std::vector<float> h_A(N * N);
    std::vector<float> h_B(N * N);
    std::vector<float> h_C(N * N);

    // Generate random values for A and B
    generateRandomMatrix(h_A.data(), N, N);
    generateRandomMatrix(h_B.data(), N, N);

    float* d_A, * d_B, * d_C;
    size_t matrixSize = N * N * sizeof(float);

    // Start total CPU timer
    hipEvent_t cpu_start, cpu_stop;
    CHECK_CUDA(hipEventCreate(&cpu_start));
    CHECK_CUDA(hipEventCreate(&cpu_stop));
    CHECK_CUDA(hipEventRecord(cpu_start, 0));

    // Allocate device memory for matrices A, B, and C
    CHECK_CUDA(hipMalloc(&d_A, matrixSize));
    CHECK_CUDA(hipMalloc(&d_B, matrixSize));
    CHECK_CUDA(hipMalloc(&d_C, matrixSize));

    // Start memcpy timer
    hipEvent_t memcpy_start, memcpy_stop;
    CHECK_CUDA(hipEventCreate(&memcpy_start));
    CHECK_CUDA(hipEventCreate(&memcpy_stop));
    CHECK_CUDA(hipEventRecord(memcpy_start, 0));

    // Copy matrices A and B to device memory
    CHECK_CUDA(hipMemcpy(d_A, h_A.data(), matrixSize, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_B, h_B.data(), matrixSize, hipMemcpyHostToDevice));

    // Stop memcpy timer
    CHECK_CUDA(hipEventRecord(memcpy_stop, 0));
    CHECK_CUDA(hipEventSynchronize(memcpy_stop));
    float memcpy_time = 0;
    CHECK_CUDA(hipEventElapsedTime(&memcpy_time, memcpy_start, memcpy_stop));

    // Create cuBLAS handle
    hipblasHandle_t handle;
    CHECK_CUBLAS(hipblasCreate(&handle));

    // Set cuBLAS to use Tensor Cores
    CHECK_CUBLAS(hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH));

    // Start GPU timer
    hipEvent_t gpu_start, gpu_stop;
    CHECK_CUDA(hipEventCreate(&gpu_start));
    CHECK_CUDA(hipEventCreate(&gpu_stop));
    CHECK_CUDA(hipEventRecord(gpu_start, 0));

    // Perform matrix multiplication: C = alpha * A * B + beta * C
    CHECK_CUBLAS(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, d_A, N, d_B, N, &beta, d_C, N));

    // Stop GPU timer
    CHECK_CUDA(hipEventRecord(gpu_stop, 0));
    CHECK_CUDA(hipEventSynchronize(gpu_stop));
    float gpu_time = 0;
    CHECK_CUDA(hipEventElapsedTime(&gpu_time, gpu_start, gpu_stop));

    // Copy result matrix C back to host memory
    CHECK_CUDA(hipMemcpy(h_C.data(), d_C, matrixSize, hipMemcpyDeviceToHost));

    // Stop total CPU timer
    CHECK_CUDA(hipEventRecord(cpu_stop, 0));
    CHECK_CUDA(hipEventSynchronize(cpu_stop));
    float total_time = 0;
    CHECK_CUDA(hipEventElapsedTime(&total_time, cpu_start, cpu_stop));

    // Clean up resources
    CHECK_CUDA(hipFree(d_A));
    CHECK_CUDA(hipFree(d_B));
    CHECK_CUDA(hipFree(d_C));
    CHECK_CUBLAS(hipblasDestroy(handle));
    CHECK_CUDA(hipEventDestroy(gpu_start));
    CHECK_CUDA(hipEventDestroy(gpu_stop));
    CHECK_CUDA(hipEventDestroy(memcpy_start));
    CHECK_CUDA(hipEventDestroy(memcpy_stop));
    CHECK_CUDA(hipEventDestroy(cpu_start));
    CHECK_CUDA(hipEventDestroy(cpu_stop));

    // Output timing results
    std::cout << "Memcpy time: " << memcpy_time << " ms" << std::endl;
    std::cout << "GPU computation time: " << gpu_time << " ms" << std::endl;
    std::cout << "Total time (CPU perspective): " << total_time << " ms" << std::endl;

    return 0;
}
